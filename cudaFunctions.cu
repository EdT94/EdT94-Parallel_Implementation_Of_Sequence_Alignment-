#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>


/* This function runs on the GPU, gets two characters and checks if they are both in the 'conservativeGroupLength' group, if so, returns 1, otherwise 0 */
__device__ int inConservative(const char seq1Char, const char seq2Char)
{
	int char1InGroup = 0 ;
	int char2InGroup = 0 ;
	const char * conservativeGroups[9] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK", "FYW", "HY", "MILF" };

 	int conservativeGroupLength = sizeof(conservativeGroups)/sizeof(conservativeGroups[0]);
	//#pragma omp parallel for
	for(int i = 0 ; i < conservativeGroupLength ; i++)
	{
		const char * group = conservativeGroups[i];
		for(int j = 0 ; j < sizeof(group)/sizeof(char) ; j++)
		{
			if(seq1Char == group[j])
				char1InGroup = 1;
			else if(seq2Char == group[j])
				char2InGroup = 1;
					
		}
		if(char1InGroup == 1 && char2InGroup == 1)
			return 1;
		else
		{
			char1InGroup = 0 ;
			char2InGroup = 0 ;
		}
	}
	
	return 0;
	
}



/* This function runs on the GPU, gets two characters and checks if they are both in the 'semiConservativeGroups' group, if so, returns 1, otherwise 0 */
__device__ int inSemiConservative(const char seq1Char,const char seq2Char)
{
	int char1InGroup = 0 ;
	int char2InGroup = 0 ;
	const char * semiConservativeGroups[11] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK", "NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };

	int semiConservativeGroupLength = sizeof(semiConservativeGroups)/sizeof(semiConservativeGroups[0]);
	//#pragma omp parallel for
	for(int i = 0 ; i < semiConservativeGroupLength ; i++)
	{
		const char * group = semiConservativeGroups[i];
		for(int j = 0 ; j < sizeof(group)/sizeof(char) ; j++)
		{
			if(seq1Char == group[j])
				char1InGroup = 1;
			else if(seq2Char == group[j])
				char2InGroup = 1;
					
		}
		if(char1InGroup == 1 && char2InGroup == 1)
			return 1;
		else
		{
			char1InGroup = 0 ;
			char2InGroup = 0 ;
		}
	}
	
	return 0;
}



/* A function that gets an array and its length and returns the index with the largest value */
int findIndexOfMaxScore(float * offset, int length)
{
	int indexOfMax = 0;
	
	for( int i = 1 ; i < length ; i ++ )
	{
		if(offset[i] > offset[indexOfMax])
			indexOfMax = i ;
		
	}
	
	return indexOfMax;
}



/* A function that runs on the GPU. Each thread whose value is less than the 'maxOffset' participates in the calculation. Each thread is responsible for specific offset, comparing one letter from seq2 to seq1. Each thread runs 'seq2Length' times and adds to it's own offset the specific weight value for each comparison between letters */
__global__  void computeScoresOnGPU(float * offset, char * seq1, char * seq2, int seq2Length, float * weights, int maxOffset)
{
	int threadNum = blockDim.x * blockIdx.x + threadIdx.x;
	offset[threadNum] = 0;
	
	if(threadNum <= maxOffset)
	{
		//#pragma omp parallel for
		for(int i = 0 ; i < seq2Length ; i++)
		{
			if(seq1[threadNum+i] == seq2[i]) 
				offset[threadNum] += weights[0];
			
				
			else if(inConservative(seq1[threadNum+i], seq2[i]) == 1)	
				offset[threadNum] += weights[1];
			
				
			else if (inSemiConservative(seq1[threadNum+i], seq2[i]) == 1) 
				offset[threadNum] += weights[2];
				 
			else 
				offset[threadNum] += weights[3];
			
		}
	}
}



/* A function that alocates memory on GPU and copies the data to GPU */
void * cudaInit(void * array, size_t size)
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
	
	void *dev_data;
	 
      
	// Allocate memory on GPU to copy the data from the host
	err = hipMalloc((void **)&dev_data, size);
	if (err != hipSuccess)
	{
		printf("Failed to allocate device memory on GPU - %s\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
   	}
    
   	 // Copy data from host to the GPU memory
   	 err = hipMemcpy(dev_data, array, size, hipMemcpyHostToDevice);
    	if (err != hipSuccess)
    	{
       	 printf( "Failed to Copy data from host to the GPU memory - %s\n", hipGetErrorString(err));
       	 exit(EXIT_FAILURE);
   	}

	return dev_data;
}



/* Free allocated memory on GPU */
void freeCuda(void * data)
{
	if (hipFree(data) != hipSuccess )
	{
		printf("Failed to free device data - %s\n", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
}



/* A function that prepares the data for calculation with cuda, and copies it back to the host*/
int computeOnGPU(float ** mutantsScore, char * seq1, char * seq2, int seq1Length, int seq2Length, float * weights,int weightsSize, int * offsetOfMaxScore)
{
	char * dev_seq1, * dev_seq2;
	float * dev_weights, * dev_offset, * mutantsTemp;
	int maxOffset = (seq1Length-seq2Length);
	int threadsPerBlock = 256 ;
	int blocksPerGrid = (maxOffset + threadsPerBlock - 1 ) / threadsPerBlock;
		
	mutantsTemp = (float*)malloc(sizeof(float)*maxOffset);
	if (mutantsTemp == NULL)
	{
		printf("Failed to allocate memory for 'mutantsTemp' in 'computeOnGPU' function\n");
		exit(0);
	}

	size_t seq1LengthCuda = seq1Length * sizeof(char);
	size_t seq2LengthCuda = seq2Length * sizeof(char);
	size_t sizeOfWeightsArrayCuda = weightsSize * sizeof(float);
	size_t sizeOfMutantsArrayCuda = maxOffset * sizeof(float);


	dev_seq1 = (char *)cudaInit((void *)seq1, seq1LengthCuda);
	dev_seq2 = (char *)cudaInit((void *)seq2, seq2LengthCuda);
	dev_weights = (float *)cudaInit((void*)weights, sizeOfWeightsArrayCuda);
	

	if (hipMalloc((void **)&dev_offset, sizeOfMutantsArrayCuda) != hipSuccess)
   	{
		printf("Failed to allocate device memory on GPU - %s\n", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
    	}
    	
	computeScoresOnGPU<<<blocksPerGrid, threadsPerBlock>>>(dev_offset, dev_seq1, dev_seq2, seq2Length, dev_weights, maxOffset);

	if (hipGetLastError() != hipSuccess)
	{
		printf("Failed to launch kernel -  %s\n", hipGetErrorString(hipGetLastError()));
		exit(EXIT_FAILURE);
	}
		
	if (hipMemcpy(mutantsTemp, dev_offset, sizeOfMutantsArrayCuda, hipMemcpyDeviceToHost) != hipSuccess)
	{
       	printf( "Failed to Copy data from GPU to host memory - %s\n", 				hipGetErrorString(hipGetLastError()));
       	exit(EXIT_FAILURE);
    	}
		 
	*offsetOfMaxScore = findIndexOfMaxScore(mutantsTemp, maxOffset);
	*mutantsScore = mutantsTemp;	
			
	freeCuda(dev_seq1);
	freeCuda(dev_seq2);
	freeCuda(dev_weights);
	freeCuda(dev_offset);
	freeCuda(dev_signs);

	return 0;
	
}










